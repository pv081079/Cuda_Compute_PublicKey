#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "secp256k1.cuh"
#include <ctime>

__global__ void generateRandomPrivateKeyKernel(u64* d_privateKey, unsigned long long seed) {
    int idx = threadIdx.x;
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    d_privateKey[idx] = hiprand(&state);
}

__global__ void computePublicKey(Point* d_publicKey, const u64* d_privateKey) {
    secp256k1PublicKey(d_publicKey, d_privateKey);
}

int main() {
    // Allocate memory on the device for private key and public key
    u64* d_privateKey;
    hipMalloc((void**)&d_privateKey, 4 * sizeof(u64));

    Point* d_publicKey;
    hipMalloc((void**)&d_publicKey, sizeof(Point));

    // Generate random seed using current time
    unsigned long long seed = static_cast<unsigned long long>(time(nullptr));

    // Generate random private key on the device
    generateRandomPrivateKeyKernel<<<1, 4>>>(d_privateKey, seed);
    hipDeviceSynchronize();

    // Launch kernel to compute public key
    computePublicKey<<<1, 1>>>(d_publicKey, d_privateKey);
    hipDeviceSynchronize();

    // Copy the result back to the host
    u64 h_privateKey[4];
    hipMemcpy(h_privateKey, d_privateKey, 4 * sizeof(u64), hipMemcpyDeviceToHost);

    Point h_publicKey;
    hipMemcpy(&h_publicKey, d_publicKey, sizeof(Point), hipMemcpyDeviceToHost);

    // Print the private key
    std::cout << "Private Key: ";
    for (int i = 3; i >= 0; --i) {
        std::cout << std::hex << std::setfill('0') << std::setw(16) << h_privateKey[i];
    }
    std::cout << std::endl;

    // Print the public key
    std::cout << "Public Key:" << std::endl;
    std::cout << "X: ";
    for (int i = 3; i >= 0; --i) {
        std::cout << std::hex << std::setfill('0') << std::setw(16) << h_publicKey.x[i];
    }
    std::cout << std::endl;

    std::cout << "Y: ";
    for (int i = 3; i >= 0; --i) {
        std::cout << std::hex << std::setfill('0') << std::setw(16) << h_publicKey.y[i];
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_privateKey);
    hipFree(d_publicKey);

    return 0;
}

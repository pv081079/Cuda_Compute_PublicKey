#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "secp256k1.cuh"
#include <ctime>

__global__ void generateRandomPrivateKeyKernel(u64* d_privateKeys, unsigned long long seed, int numKeys) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numKeys * 4) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);

        d_privateKeys[idx] = hiprand(&state);
    }
}

__global__ void computePublicKey(Point* d_publicKeys, const u64* d_privateKeys, int numKeys) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numKeys) {
        secp256k1PublicKey(&d_publicKeys[idx], &d_privateKeys[idx * 4]);
    }
}

void checkCudaError(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << message << ": " << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    // Query the GPU for its properties
    hipDeviceProp_t prop;
    checkCudaError(hipGetDeviceProperties(&prop, 0), "Getting device properties");

    std::cout << "GPU Model: " << prop.name << std::endl;
    std::cout << "Number of SMs: " << prop.multiProcessorCount << std::endl;
    std::cout << "Max Threads per Block: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Max Threads per SM: " << prop.maxThreadsPerMultiProcessor << std::endl;

    int maxThreadsPerBlock = prop.maxThreadsPerBlock;
    int maxKeys = prop.multiProcessorCount * (prop.maxThreadsPerMultiProcessor / 256) * 256;  // Estimate based on the GPU's capabilities
    std::cout << "Generating maximum of " << maxKeys << " private keys and corresponding public keys." << std::endl;

    int threadsPerBlock = 256;  // Safe value to avoid resource limit issues
    int blocksForKeys = (maxKeys + threadsPerBlock - 1) / threadsPerBlock;
    int blocksForPrivateKey = (maxKeys * 4 + threadsPerBlock - 1) / threadsPerBlock;

    // Allocate memory on the device for private keys and public keys
    u64* d_privateKeys;
    checkCudaError(hipMalloc((void**)&d_privateKeys, maxKeys * 4 * sizeof(u64)), "Allocating device memory for private keys");

    Point* d_publicKeys;
    checkCudaError(hipMalloc((void**)&d_publicKeys, maxKeys * sizeof(Point)), "Allocating device memory for public keys");

    // Generate random seed using current time
    unsigned long long seed = static_cast<unsigned long long>(time(nullptr));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "Creating start event");
    checkCudaError(hipEventCreate(&stop), "Creating stop event");

    // Start timing
    checkCudaError(hipEventRecord(start), "Recording start event");

    // Generate random private keys on the device
    generateRandomPrivateKeyKernel<<<blocksForPrivateKey, threadsPerBlock>>>(d_privateKeys, seed, maxKeys);
    checkCudaError(hipGetLastError(), "Launching generateRandomPrivateKeyKernel");
    checkCudaError(hipDeviceSynchronize(), "Synchronizing after generateRandomPrivateKeyKernel");

    // Launch kernel to compute public keys
    computePublicKey<<<blocksForKeys, threadsPerBlock>>>(d_publicKeys, d_privateKeys, maxKeys);
    checkCudaError(hipGetLastError(), "Launching computePublicKey");
    checkCudaError(hipDeviceSynchronize(), "Synchronizing after computePublicKey");

    // Stop timing
    checkCudaError(hipEventRecord(stop), "Recording stop event");
    checkCudaError(hipEventSynchronize(stop), "Synchronizing stop event");

    // Calculate elapsed time
    float milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop), "Calculating elapsed time");

    // Calculate keys per second
    float seconds = milliseconds / 1000.0f;
    float keysPerSecond = maxKeys / seconds;

    std::cout << "Generated " << maxKeys << " private keys and public keys in " << seconds << " seconds." << std::endl;
    std::cout << "Keys per second: " << keysPerSecond << std::endl;

    // Free device memory
    checkCudaError(hipFree(d_privateKeys), "Freeing device memory for private keys");
    checkCudaError(hipFree(d_publicKeys), "Freeing device memory for public keys");

    // Destroy CUDA events
    checkCudaError(hipEventDestroy(start), "Destroying start event");
    checkCudaError(hipEventDestroy(stop), "Destroying stop event");

    return 0;
}
